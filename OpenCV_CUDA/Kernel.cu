#include "hip/hip_runtime.h"
// opencv1.cpp : Defines the entry point for the console application.
//
#include <stdio.h>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
using namespace cv;

using namespace std;



__global__
void add(int n, int *x, int *y)
{

  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
  {
	  y[i] = y[i]/3 +5;
	  x[i] = x[i] + 5;
  }

}


#pragma warning (disable : 4996)
int main(int argc, char* argv[])
{
	VideoCapture cap;
	//	// open the default camera, use something different from 0 otherwise;
	//	// Check VideoCapture documentation.
		if (!cap.open(0))
			return 0;

	for (;;)
	{
		Mat image;
		cap >> image;
		if (image.empty()) break; // end of video stream
		imshow("this is you, smile! :)", image);
		

	//Mat image;

	////The second parameter has been used to change the default return image
	//image = imread("C://Users//Apratim//source//repos//OpenCV//x64//Debug//data//lena.jpg", IMREAD_COLOR);

	//// if the argument is < 0 it returns the original image. 

	//if (image.empty())
	//{
	//	return -1;
	//}

	//namedWindow("Display window", WINDOW_AUTOSIZE); // Create a window for display.
	//												//! [window]
	//									//! [imshow]
	//imshow("Display window", image);                // Show our image inside it.
													//! [imshow]
		uint8_t* pixelPtr = (uint8_t*)image.data;
		int cn = image.channels();
		Scalar_<int> bgrPixel;

		//int *bPixelVal, *gPixelVal;
		int size = image.rows * image.cols;

		int *x, *y;

		// Allocate Unified Memory � accessible from CPU or GPU
		hipMallocManaged(&x, size * sizeof(int));
		hipMallocManaged(&y, size * sizeof(int));

		/*x = bPixelVal;
		y = gPixelVal;
	*/

		int c = 0;
		for (int i = 0; i < image.rows; i++)
		{
			for (int j = 0; j < image.cols; j++)
			{
				bgrPixel.val[0] = pixelPtr[i*image.cols*cn + j*cn + 0]; // B
				bgrPixel.val[1] = pixelPtr[i*image.cols*cn + j*cn + 1]; // G
				bgrPixel.val[2] = pixelPtr[i*image.cols*cn + j*cn + 2]; // R

				x[c] = pixelPtr[i*image.cols*cn + j*cn + 0]; // B
				y[c] = pixelPtr[i*image.cols*cn + j*cn + 1]; // G



				c = c++;

				// do something with BGR values...
			}
		}



		// Run kernel on 50K elements on the GPU
		add << <1, 1024 >> > (size, x, y);

		// Wait for GPU to finish before accessing on host
		hipDeviceSynchronize();

		c = 0;
		for (int i = 0; i < image.rows; i++)
		{
			for (int j = 0; j < image.cols; j++)
			{
				image.at<Vec3b>(i, j)[0] = x[c];
				image.at<Vec3b>(i, j)[1] = y[c];
				//bPixelVal[c] = pixelPtr[i*image.cols*cn + j*cn + 2]; // R

				c = c++;

				// do something with BGR values...
			}
		}


		namedWindow("Output");
		imshow("Output", image);

		//cvWaitKey(0);

		
		// Free memory
		hipFree(x);
		hipFree(y);
		if (waitKey(10) == 27) break; // stop capturing by pressing ESC 
	}
	return 0;
}
